#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int numberOfThreads = 256;
const double pi = 3.14159265358979323846;

__global__ void intssss(int N, 
			int *primIndices_d,
			int *contIndices_d,
			double *exponents_d,
			double *primNormalization_d,
			double *coefficients_d,
			int *contCounter_d,
			int *contLength_d,
			double *origin_d,
			int *angularMoments_d,
			double *integralValues_d,
			int control,
			int kernelIter)
{
  int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;
  int blockIndex = blockIdx.x;
  int global1 = threadIndex + blockIndex*blockDim.x*blockDim.y;
  int global = global1 + kernelIter; 
  
  int aa, bb, rr, ss, ii, jj, kk, ll;
  int contractionID;
  double exponentII, exponentJJ, exponentKK, exponentLL;
  double coefficientsII, coefficientsJJ, coefficientsKK, coefficientsLL;
  double primNormII, primNormJJ, primNormKK, primNormLL;
  int exponentIterII, exponentIterJJ, exponentIterKK, exponentIterLL;
  double IIx, IIy, IIz, JJx, JJy, JJz, KKx, KKy, KKz, LLx, LLy, LLz;
  double preIntegral, normIntegral; 
  double etha;
  int lAA, lBB, lRR, lSS; // Angular moments of contractions
  int integralCase;

  double A, B, C, D, KIJ, KKL, rPx, rPy, rPz, rQx, rQy, rQz, rPQ, rIJ, rKL, tFunc, tFuncsqrt, F, prefact;

  if(global1< control)
    {
      // ID of unic integrals
      contractionID = primIndices_d[global*5];

      // Contraction Indices
      aa = contIndices_d[contractionID*4];
      bb = contIndices_d[contractionID*4+1];
      rr = contIndices_d[contractionID*4+2];
      ss = contIndices_d[contractionID*4+3];
      // Primitive indices
      ii = primIndices_d[global*5+1];
      jj = primIndices_d[global*5+2];
      kk = primIndices_d[global*5+3];
      ll = primIndices_d[global*5+4];
      
      lAA = angularMoments_d[aa-1];
      lBB = angularMoments_d[bb-1];
      lRR = angularMoments_d[rr-1];
      lSS = angularMoments_d[ss-1];
      
      exponentIterII = contCounter_d[aa-1] + ii - 1;
      exponentIterJJ = contCounter_d[bb-1] + jj - 1;
      exponentIterKK = contCounter_d[rr-1] + kk - 1;
      exponentIterLL = contCounter_d[ss-1] + ll - 1;

      exponentII = exponents_d[exponentIterII];
      exponentJJ = exponents_d[exponentIterJJ];
      exponentKK = exponents_d[exponentIterKK];
      exponentLL = exponents_d[exponentIterLL];

      coefficientsII = coefficients_d[exponentIterII];
      coefficientsJJ = coefficients_d[exponentIterJJ];
      coefficientsKK = coefficients_d[exponentIterKK];
      coefficientsLL = coefficients_d[exponentIterLL];

      primNormII = primNormalization_d[exponentIterII];
      primNormJJ = primNormalization_d[exponentIterJJ];
      primNormKK = primNormalization_d[exponentIterKK];
      primNormLL = primNormalization_d[exponentIterLL];

      IIx = origin_d[(aa*3)-3];
      IIy = origin_d[(aa*3)-2];
      IIz = origin_d[(aa*3)-1];
      JJx = origin_d[(bb*3)-3];
      JJy = origin_d[(bb*3)-2];
      JJz = origin_d[(bb*3)-1];
      KKx = origin_d[(rr*3)-3];
      KKy = origin_d[(rr*3)-2];
      KKz = origin_d[(rr*3)-1];
      LLx = origin_d[(ss*3)-3];
      LLy = origin_d[(ss*3)-2];
      LLz = origin_d[(ss*3)-1];
    
      A = exponentII + exponentJJ;
      B = exponentKK + exponentLL;
      C = exponentII*exponentJJ;
      D = exponentKK*exponentLL;

      etha = (A*B)/(A+B);

      rIJ = (IIx-JJx)*(IIx-JJx) + (IIy-JJy)*(IIy-JJy) + (IIz-JJz)*(IIz-JJz);
      rKL = (KKx-LLx)*(KKx-LLx) + (KKy-LLy)*(KKy-LLy) + (KKz-LLz)*(KKz-LLz);

      KIJ = exp(-(C/A)*rIJ);
      KKL = exp(-(D/B)*rKL);

      prefact = (pi*pi*pi)/(A*B*(sqrt(A+B)));

      rPx =(exponentII*IIx+exponentJJ*JJx)/A;
      rPy =(exponentII*IIy+exponentJJ*JJy)/A;
      rPz =(exponentII*IIz+exponentJJ*JJz)/A;
      rQx = (exponentKK*KKx+exponentLL*LLx)/B;
      rQy = (exponentKK*KKy+exponentLL*LLy)/B;
      rQz = (exponentKK*KKz+exponentLL*LLz)/B;
      
      rPQ = (rPx-rQx)*(rPx-rQx) + (rPy-rQy)*(rPy-rQy) + (rPz-rQz)*(rPz-rQz);

      tFunc = etha*rPQ;

      tFuncsqrt = sqrt(tFunc);

      if(tFunc == 0.0)
	F = 2/(sqrt(pi));
      else
	F = erf(tFuncsqrt)/tFuncsqrt;
      
      integralCase = 64*lAA + 16*lBB + 4*lRR + lSS;

      switch(integralCase)
	{
	case 0: // Integral (s,s|s,s)
	  preIntegral = prefact*KIJ*KKL*F;
	  break;
	case 1: // Integral (s,s|s,p)
	  preIntegral = (2*(exponentKK*((KKx-LLx)+(rPx-rQx))*etha)*(exponentKK*((KKy-LLy)+(rPy-rQy))*etha)*(exponentKK*((KKz-LLz)+(rPz-rQz))*etha)*F)/(B*B*B);
	  break;
	case 4: // Integral (s,s|p,s)
	  preIntegral = -(2*(exponentLL*((KKx-LLx)+(rPx-rQx))*etha)*(exponentLL*((KKy-LLy)+(rPy-rQy))*etha)*(exponentLL*((KKz-LLz)+(rPz-rQz))*etha)*F)/(B*B*B);
	  break;
	case 16: // Integral (s,p|s,s)
	  preIntegral = (2*(exponentII*((IIx-JJx)+(rPx-rQx))*etha)*(exponentII*((IIy-JJy)+(rPy-rQy))*etha)*(exponentII*((IIz-JJz)+(rPz-rQz))*etha)*F)/(A*A*A);
	  break;
	case 64: // Integral (p,s|s,s)
	  preIntegral = (2*(exponentJJ*((IIx-JJx)+(rPx-rQx))*etha)*(exponentJJ*((IIy-JJy)+(rPy-rQy))*etha)*(exponentJJ*((IIz-JJz)+(rPz-rQz))*etha)*F)/(A*A*A);
	  break;
	case 5: // Integral (s,s|p,p)
	  preIntegral = (1/(4*pow(B,6))*(B+2*exponentKK*(KKx-LLx)*(-exponentLL*((KKx-LLx)+(rPx-rQx))*etha)+etha*(-1-2*exponentLL*(KKx-LLx)*(rPx-rQx)+2*(rPx-rQx)*(rPx-rQx)*etha))*(4*exponentKK*exponentKK*(KKy-LLy)*(KKz-LLz)*(exponentLL*((KKy-LLy)-(rPy-rQy))*etha)*(exponentLL*((KKz-LLz)-(rPz-rQz))*etha)+(B+etha*(-1-2*exponentLL*(KKy-LLy)*(rPy-rQy)+2*(rPy-rQy)*(rPy-rQy)*etha))*(B+etha*(-1-2*exponentLL*(KKz-LLz)*(rPz-rQz)+2*(rPz-rQz)*(rPz-rQz)*etha))+2*exponentKK*(2*exponentLL*exponentLL*(KKy-LLy)*(KKz-LLz)*((KKz-LLz)*(rPy-rQy)+(KKy-LLy)*(rPz-rQz))*etha-exponentLL*(4*(KKy-LLy)*(KKz-LLz)*(rPy-rQy)*(rPz-rQz)*etha*etha+(KKz-LLz)*(KKz-LLz)*(B+etha*(-1+2*(rPy-rQy)*(rPy-rQy)*etha))+(KKy-LLy)*(KKy-LLy)*(B+etha*(-1+2*(rPz-rQz)*(rPz-rQz)*etha)))+etha*((KKz-LLz)*(rPz-rQz)*(B+etha*(-1+2*(rPy-rQy)*(rPy-rQy)*etha))+(KKy-LLy)*(rPy-rQy)*(B+etha*(-1+2*(rPz-rQz)*(rPz-rQz)*etha))))))*F;
	  break;
	case 17: // Integral (s,p|s,p)
	  preIntegral = (1/(4*A*A*A*A*A*B*B*B*B*exponentKK*(rPx-rQx)*exponentII))*(2*(IIx-JJx)*A*A*B*((KKx-LLx)*(rPx-rQx)+exponentKK*etha)+A*A*B*(rPx-rQx)*exponentII*(exponentKK-2*(KKx-LLx)*(rPx-rQx)-2*exponentKK*(rPx-rQx)*(rPx-rQx)*etha))*(-2*exponentII*(IIy-JJy)*(exponentKK*(KKy-LLy)+(rPy-rQy)*etha)+etha*(-1+2*exponentKK*(KKy-LLy)*(rPy-rQy)+2*(rPy-rQy)*(rPy-rQy)*etha))*(-2*exponentII*(IIz-JJz)*(exponentKK*(KKz-LLz)+(rPz-rQz)*etha)+etha*(-1+2*exponentKK*(KKz-LLz)*(rPz-rQz)+2*(rPz-rQz)*(rPz-rQz)*etha))*F;
	  break;
	case 20: // Integral (s,p|p,s)
	  preIntegral = (1/(4*A*A*A*A*A*B*B*B*B*exponentLL*(rPx-rQx)*exponentII))*(2*(IIx-JJx)*A*A*B*(-(KKx-LLx)*(rPx-rQx)+exponentLL*etha)+A*A*B*(rPx-rQx)*exponentII*(exponentLL+2*(KKx-LLx)*(rPx-rQx)-2*exponentLL*(rPx-rQx)*(rPx-rQx)*etha))*(2*exponentII*(IIy-JJy)*(exponentLL*(KKy-LLy)-(rPy-rQy)*etha)+etha*(-1-2*exponentLL*(KKy-LLy)*(rPy-rQy)+2*(rPy-rQy)*(rPy-rQy)*etha))*(-2*exponentII*(IIz-JJz)*(exponentLL*(KKz-LLz)-(rPz-rQz)*etha)+etha*(-1-2*exponentLL*(KKz-LLz)*(rPz-rQz)+2*(rPz-rQz)*(rPz-rQz)*etha))*F;
	  break;
	case 65: // Integral (p,s|s,p)
	  preIntegral = -(1/(4*A*A*A*A*A*B*B*B*B*exponentKK*(rPx-rQx)*exponentJJ))*(2*exponentJJ*(IIy-JJy)*(exponentKK*(KKy-LLy)+(rPy-rQy)*etha)+etha*(-1+2*exponentKK*(KKy-LLy)*(rPy-rQy)+2*(rPy-rQy)*(rPy-rQy)*etha))*(2*exponentJJ*(IIz-JJz)*(exponentKK*(KKz-LLz)+(rPz-rQz)*etha)+etha*(-1+2*exponentKK*(KKz-LLz)*(rPz-rQz)+2*(rPz-rQz)*(rPz-rQz)*etha))*(2*(IIx-JJx)*A*A*B*((KKx-LLx)*(rPx-rQx)+exponentKK*etha)+A*A*B*(rPx-rQx)*exponentJJ*etha*(2*(KKx-LLx)*(rPx-rQx)+exponentKK*(-1+2*(rPx-rQx)*(rPx-rQx)*etha)))*F;
	  break;
	case 68: // Integral (p,s|p,s)
	  preIntegral = -1/(4*A*A*A*A*exponentLL*A*(rPx-rQx)*B*B*B*B*exponentJJ)*(2*(IIx-JJx)*A*B*((rPx-rQx)*A*(KKx-LLx)-A*exponentLL*etha)+A*(KKx-LLx)*B*exponentJJ*etha*(A*exponentLL+2*(KKx-LLx)*A*(rPx-rQx)-2*A*exponentLL*(rPx-rQx)*(rPx-rQx)*etha))*(exponentJJ*(-2*exponentLL*(IIx-JJx)*(KKx-LLx)+2*(IIy-JJy)*(rPy-rQy)*etha)+etha*(-1-2*exponentLL*(KKy-LLy)*(rPy-rQy)+2*(rPy-rQy)*(rPy-rQy)*etha))*(2*exponentJJ*(KKz-LLz)*(exponentLL*(KKz-LLz)-(rPz-rQz)*etha)+etha*(1+2*exponentLL*(KKz-LLz)*(rPz-rQz)-2*(rPz-rQz)*(rPz-rQz)*etha))*F;
	  break;
	case 80: // Integral (p,p|s,s)
	  preIntegral = 1/(4*A*A*A*A*A*A)*(A-2*exponentII*(IIx-JJx)*(exponentJJ*(IIx-JJx)+(rPx-rQx)*etha)+etha*(2*exponentJJ*(IIx-JJx)*(rPx-rQx)-1+2*(rPx-rQx)*(rPx-rQx)*etha)*(4*exponentII*exponentII*(IIy-JJy)*(IIz-JJz)*(exponentJJ*(IIy-JJy)+(rPy-rQy)*etha)*(exponentJJ*(IIz-JJz)+(rPx-rQx)*etha)+(A+etha*(-1+2*exponentJJ*(IIy-JJy)*(rPy-rQy)+2*(rPy-rQy)*(rPy-rQy)))*(A+etha*(-1+2*exponentJJ*(IIz-JJz)*(rPz-rQz)+2*(rPz-rQz)*(rPy-rQy)))-2*exponentII*(2*exponentJJ*exponentJJ*(IIy-JJy)*(IIz-JJz)*((IIz-JJz)*(rPy-rQy)+(rPz-rQz)*((IIy-JJy))*etha)+exponentJJ*(4*(IIy-JJy)*(IIz-JJz)*(rPy-rQy)*(rPz-rQz)*etha*etha+(IIz-JJz)*(IIz-JJz)*(A+etha*(-1+2*(rPy-rQy)*(rPy-rQy)*etha))+(IIy-JJy)*(IIy-JJy)*(A+etha*(-1+2*(rPz-rQz)*(rPz-rQz)*etha)))+etha*((IIz-JJz)*(rPz-rQz)*(A+etha*(-1+2*(rPz-rQz)*(rPy-rQy)*etha))+(IIy-JJy)*(rPy-rQy)*(A+etha*(-1+2*(rPz-rQz)*(rPy-rQy)*etha))))))*F;
	  break;
	case 21: // Integral (s,p|p,p)
	  preIntegral = -1/(4*A*A*A*A*A*B*B*B*B*B*B*exponentKK*exponentLL)*(exponentII*(IIy-JJy)*(-B+2*exponentKK*(KKy-LLy)*(-(rPy-rQy)*(etha)+exponentLL*(KKy-LLy))+etha*(-2*(rPy-rQy)*(rPy-rQy)*etha+1+2*exponentLL*(KKy-LLy)*(rPy-rQy))+etha*(exponentLL*((KKy-LLy)-2*(KKy-LLy)*(rPy-rQy)*(rPy-rQy)*etha)-(KKy-LLy)*exponentKK*(1+2*exponentLL*(KKy-LLy)*(rPy-rQy)-2*(rPy-rQy)*(rPy-rQy)*etha)+(rPy-rQy)*(B+etha*(2*(rPy-rQy)*(rPy-rQy)*etha-3)))))*(exponentII*(IIz-JJz)*(-B+2*exponentKK*(KKz-LLz)*(-etha*(rPz-rQz)+(KKz-LLz)*exponentLL)+etha*(-2*(rPz-rQz)*(rPz-rQz)+1+2*exponentLL*(KKz-LLz)*(rPz-rQz)))+etha*(exponentLL*((KKz-LLz)-2*(KKz-LLz)*(rPz-rQz)*(rPz-rQz)*etha)-exponentKK*(KKz-LLz)*(1+2*exponentLL*(KKz-LLz)*(rPz-rQz)-2*(rPz-rQz)*(rPz-rQz)*etha)+(rPz-rQz)*(B+etha*(2*etha*(rPz-rQz)*(rPz-rQz)-3))))*(A*exponentLL*etha*(exponentKK*(KKx-LLx)*A*exponentKK*(-1+2*(rPx-rQx)*(rPx-rQx))+A*exponentKK*(rPx-rQx)*(B+etha*(2*(rPx-rQx)*(rPx-rQx)*etha-3))+exponentLL*(KKx-LLx)*(A*exponentKK-2*(KKx-LLx)*A*(rPx-rQx)-2*exponentKK*A*(rPx-rQx)*(rPx-rQx)))+exponentII*(IIx-JJx)*(2*exponentLL*(KKx-LLx)*(KKx-LLx)*A*A*exponentLL-2*(KKx-LLx)*A*A*exponentLL*(rPx-rQx)*etha+A*exponentKK*(2*(KKx-LLx)*A*(rPx-rQx)*etha+A*exponentLL*(etha-B-2*(rPx-rQx)*(rPx-rQx)*etha*etha))))*F;
	  break;
	case 69: // Integral (p,s|p,p)
	  preIntegral = 1/(4*A*A*A*A*A*B*B*B*B*B*B*exponentKK*exponentLL)*(exponentJJ*(IIy-JJy)*(B+2*exponentKK*(KKy-LLy)*((rPy-rQy)*(etha)-exponentLL*(KKy-LLy))+etha*(2*(rPy-rQy)*(rPy-rQy)*etha-1-2*exponentLL*(KKy-LLy)*(rPy-rQy))+etha*(exponentLL*((KKy-LLy)-2*(KKy-LLy)*(rPy-rQy)*(rPy-rQy)*etha)-(KKy-LLy)*exponentKK*(1+2*exponentLL*(KKy-LLy)*(rPy-rQy)-2*(rPy-rQy)*(rPy-rQy)*etha)+(rPy-rQy)*(B+etha*(2*(rPy-rQy)*(rPy-rQy)*etha-3)))))*(exponentJJ*(IIz-JJz)*(B+2*exponentKK*(KKz-LLz)*(etha*(rPz-rQz)-(KKz-LLz)*exponentLL)+etha*(2*(rPz-rQz)*(rPz-rQz)-1-2*exponentLL*(KKz-LLz)*(rPz-rQz)))+etha*(exponentLL*((KKz-LLz)-2*(KKz-LLz)*(rPz-rQz)*(rPz-rQz)*etha)-exponentKK*(KKz-LLz)*(1+2*exponentLL*(KKz-LLz)*(rPz-rQz)-2*(rPz-rQz)*(rPz-rQz)*etha)+(rPz-rQz)*(B+etha*(2*etha*(rPz-rQz)*(rPz-rQz)-3))))*(A*exponentLL*etha*(exponentKK*(KKx-LLx)*A*exponentKK*(1-2*(rPx-rQx)*(rPx-rQx))-A*exponentKK*(rPx-rQx)*(B+etha*(2*(rPx-rQx)*(rPx-rQx)*etha-3))+exponentLL*(KKx-LLx)*(2*(KKx-LLx)*A*(rPx-rQx)+exponentKK*A*(2*(rPx-rQx)*(rPx-rQx)*etha-1)))+exponentJJ*(IIx-JJx)*(2*exponentLL*(KKx-LLx)*(KKx-LLx)*A*A*exponentLL-2*(KKx-LLx)*A*A*exponentLL*(rPx-rQx)*etha+A*exponentKK*(2*(KKx-LLx)*A*(rPx-rQx)*etha+A*exponentLL*(etha-B-2*(rPx-rQx)*(rPx-rQx)*etha*etha))))*F;
	  break;
	case 81: // Integral (p,p|s,p)
	  preIntegral = 1/(4*A*A*A*A*A*A*B*B*B*B*B*exponentII*exponentII*exponentJJ*exponentJJ)*(-2*exponentJJ*exponentJJ*(IIx-JJx)*(IIx-JJx)*B*B*(exponentKK*(KKx-LLx) + (rPx-rQx)*etha) + exponentKK*(KKx-LLx)*(A*B*B*exponentII*exponentJJ + etha*B*B*(-exponentII*exponentJJ + 2*(IIx-JJx)*exponentII*(rPx-rQx) - 2*(IIx-JJx)*exponentJJ*(rPx-rQx) + 2*exponentII*exponentJJ*(rPx-rQx)*(rPx-rQx)*etha)) + etha*((IIx-JJx)*B*B*(exponentII-exponentJJ)*(-1 + 2*(rPx-rQx)*(rPx-rQx)*etha) + B*B*exponentII*exponentJJ*(rPx-rQx)*(A + etha*(-3 + 2*(rPx-rQx)*(rPx-rQx)*etha))))*(-2*exponentJJ*exponentJJ*(IIy-JJy)*(IIy-JJy)*B*B*(exponentKK*(KKy-LLy) + (rPy-rQy)*etha) + exponentKK*(KKy-LLy)*(A*B*B*exponentII*exponentJJ + etha*B*B*(-exponentII*exponentJJ + 2*(IIy-JJy)*exponentII*(rPy-rQy) - 2*(IIy-JJy)*exponentJJ*(rPy-rQy) + 2*exponentII*exponentJJ*(rPy-rQy)*(rPy-rQy)*etha)) + etha*((IIy-JJy)*B*B*(exponentII-exponentJJ)*(-1 + 2*(rPy-rQy)*(rPy-rQy)*etha) + B*B*exponentII*exponentJJ*(rPy-rQy)*(A + etha*(-3 + 2*(rPy-rQy)*(rPy-rQy)*etha))))*(exponentKK*(KKz-LLz)*(A + etha*(-1 + 2*exponentJJ*(IIz-JJz)*(rPz-rQz) + 2*(rPz-rQz)*(rPz-rQz)*etha)) - exponentII*(IIz-JJz)*(2*exponentJJ*(IIz-JJz)*(exponentKK*(KKz-LLz) + (rPz-rQz)*etha) + etha*(-1 + 2*exponentKK*(KKz-LLz)*(rPz-rQz) + 2*(rPz-rQz)*(rPz-rQz)*etha)) + etha*(exponentJJ*(IIz-JJz)*(-1 + 2*(rPz-rQz)*(rPz-rQz)*etha) + (rPz-rQz)*(A + etha*(-3 + 2*(rPz-rQz)*(rPz-rQz)*etha))))*F;
	  break;
	case 84: // Integral (p,p|p,s)
	  preIntegral = -1/(4*A*A*A*A*A*A*B*B*B*B*B*exponentII*exponentII*exponentJJ*exponentJJ)*(2*exponentJJ*exponentJJ*(IIx-JJx)*(IIx-JJx)*B*B*(exponentLL*(KKx-LLx)-(rPx-rQx)*etha)+etha*((IIx-JJx)*B*B*(exponentII-exponentJJ)*(2*(rPx-rQx)*(rPx-rQx)*etha-1)+B*B*exponentII*exponentJJ*(rPx-rQx)*(A+etha*(2*(rPx-rQx)*(rPx-rQx)*etha-3)))+exponentLL*(KKx-LLx)*(etha*(2*(IIx-JJx)*B*B*exponentJJ*(rPx-rQx)-A*B*B*exponentII*exponentJJ+B*B*exponentII*(exponentJJ-2*(IIx-JJx)*(rPx-rQx)-2*exponentJJ*(rPx-rQx)*(rPx-rQx)*etha))))*(2*exponentJJ*exponentJJ*(IIy-JJy)*(IIy-JJy)*B*B*(exponentLL*(KKy-LLy)-(rPy-rQy)*etha)+etha*((IIy-JJy)*B*B*(exponentII-exponentJJ)*(2*(rPy-rQy)*(rPy-rQy)*etha-1)+B*B*exponentII*exponentJJ*(rPy-rQy)*(A+etha*(2*(rPy-rQy)*(rPy-rQy)*etha-3)))+exponentLL*(KKy-LLy)*(etha*(2*(IIy-JJy)*B*B*exponentJJ*(rPy-rQy)-A*B*B*exponentII*exponentJJ+B*B*exponentII*(exponentJJ-2*(IIy-JJy)*(rPy-rQy)-2*exponentJJ*(rPy-rQy)*(rPy-rQy)*etha))))*(exponentLL*(KKz-LLz)*(A+etha*(2*exponentJJ*(IIz-JJz)*(rPz-rQz)+2*(rPz-rQz)*(rPz-rQz)*etha-1))-exponentII*(IIz-JJz)*(2*exponentJJ*(IIz-JJz)*(exponentLL*(KKz-LLz)-(rPz-rQz)*etha)+etha*(1+2*exponentLL*(KKz-LLz)*(rPz-rQz)-2*(rPz-rQz)*(rPz-rQz)*etha))+etha*(exponentJJ*((IIz-JJz)-2*(IIz-JJz)*(rPz-rQz)*(rPz-rQz)*etha)-(rPz-rQz)*(A+etha*(2*(rPz-rQz)*(rPz-rQz)*etha-3))))*F;
	  break;
	case 85: // Integral (p,p|p,p)
	  preIntegral = 1/(32*A*A*A*A*A*A*B*B*B*B*B*B)*(A*B + 2*exponentJJ*exponentLL*(IIx-JJx)*(KKx-LLx)*etha - A*etha-B*etha - 2*exponentLL*(KKx-LLx)*A*(rPx-rQx)*etha + 2*exponentJJ*(IIx-JJx)*B*(rPx-rQx)*etha + 3*etha*etha - 6*exponentJJ*(IIx-JJx)*(rPx-rQx)*etha*etha + 6*exponentLL*(KKx-LLx)*(rPx-rQx)*etha*etha - 4*exponentJJ*exponentLL*(IIx-JJx)*(KKx-LLx)*(rPx-rQx)*(rPx-rQx)*etha*etha + 2*A*(rPx-rQx)*(rPx-rQx)*etha*etha + 2*B*(rPx-rQx)*(rPx-rQx)*etha*etha - 12*(rPx-rQx)*(rPx-rQx)*etha*etha*etha + 4*exponentJJ*(IIx-JJx)*(rPx-rQx)*(rPx-rQx)*(rPx-rQx)*etha*etha*etha - 4*exponentLL*(KKx-LLx)*(rPx-rQx)*(rPx-rQx)*(rPx-rQx)*etha*etha*etha + 4*(rPx-rQx)*(rPx-rQx)*(rPx-rQx)*(rPx-rQx)*etha*etha*etha*etha - 2*exponentKK*(KKx-LLx)*(exponentLL*(KKx-LLx)*(A+etha*(-1 + 2*exponentJJ*(IIx-JJx)*(rPx-rQx) + 2*(rPx-rQx)*(rPx-rQx)*etha)) + etha*(exponentJJ*((IIx-JJx) - 2*(IIx-JJx)*(rPx-rQx)*(rPx-rQx)*etha) - (rPx-rQx)*(A + etha*(-3 + 2*(rPx-rQx)*(rPx-rQx)*etha)))) + 2*exponentII*(IIx-JJx)*(exponentJJ*(IIx-JJx)*(-B + 2*exponentKK*(KKx-LLx)*(exponentLL*(KKx-LLx) - (rPx-rQx)*etha) + etha*(1 + 2*exponentLL*(KKx-LLx)*(rPx-rQx) - 2*(rPx-rQx)*(rPx-rQx)*etha)) + etha*(exponentKK*(KKx-LLx)*(1 + 2*exponentLL*(KKx-LLx)*(rPx-rQx) - 2*(rPx-rQx)*(rPx-rQx)*etha) + exponentLL*(KKx-LLx)*(-1 + 2*(rPx-rQx)*(rPx-rQx)*etha) - (rPx-rQx)*(B + etha*(-3 + 2*(rPx-rQx)*(rPx-rQx)*etha)))))*(A*B + 2*exponentJJ*exponentLL*(IIy-JJy)*(KKy-LLy)*etha - A*etha-B*etha - 2*exponentLL*(KKy-LLy)*A*(rPy-rQy)*etha + 2*exponentJJ*(IIy-JJy)*B*(rPy-rQy)*etha + 3*etha*etha - 6*exponentJJ*(IIy-JJy)*(rPy-rQy)*etha*etha + 6*exponentLL*(KKy-LLy)*(rPy-rQy)*etha*etha - 4*exponentJJ*exponentLL*(IIy-JJy)*(KKy-LLy)*(rPy-rQy)*(rPy-rQy)*etha*etha + 2*A*(rPy-rQy)*(rPy-rQy)*etha*etha + 2*B*(rPy-rQy)*(rPy-rQy)*etha*etha - 12*(rPy-rQy)*(rPy-rQy)*etha*etha*etha + 4*exponentJJ*(IIy-JJy)*(rPy-rQy)*(rPy-rQy)*(rPy-rQy)*etha*etha*etha - 4*exponentLL*(KKy-LLy)*(rPy-rQy)*(rPy-rQy)*(rPy-rQy)*etha*etha*etha + 4*(rPy-rQy)*(rPy-rQy)*(rPy-rQy)*(rPy-rQy)*etha*etha*etha*etha - 2*exponentKK*(KKy-LLy)*(exponentLL*(KKy-LLy)*(A+etha*(-1 + 2*exponentJJ*(IIy-JJy)*(rPy-rQy) + 2*(rPy-rQy)*(rPy-rQy)*etha)) + etha*(exponentJJ*((IIy-JJy) - 2*(IIy-JJy)*(rPy-rQy)*(rPy-rQy)*etha) - (rPy-rQy)*(A + etha*(-3 + 2*(rPy-rQy)*(rPy-rQy)*etha)))) + 2*exponentII*(IIy-JJy)*(exponentJJ*(IIy-JJy)*(-B + 2*exponentKK*(KKy-LLy)*(exponentLL*(KKy-LLy) - (rPy-rQy)*etha) + etha*(1 + 2*exponentLL*(KKy-LLy)*(rPy-rQy) - 2*(rPy-rQy)*(rPy-rQy)*etha)) + etha*(exponentKK*(KKy-LLy)*(1 + 2*exponentLL*(KKy-LLy)*(rPy-rQy) - 2*(rPy-rQy)*(rPy-rQy)*etha) + exponentLL*(KKy-LLy)*(-1 + 2*(rPy-rQy)*(rPy-rQy)*etha) - (rPy-rQy)*(B + etha*(-3 + 2*(rPy-rQy)*(rPy-rQy)*etha)))))*(A*B + 2*exponentJJ*exponentLL*(IIz-JJz)*(KKz-LLz)*etha - A*etha-B*etha - 2*exponentLL*(KKz-LLz)*A*(rPz-rQz)*etha + 2*exponentJJ*(IIz-JJz)*B*(rPz-rQz)*etha + 3*etha*etha - 6*exponentJJ*(IIz-JJz)*(rPz-rQz)*etha*etha + 6*exponentLL*(KKz-LLz)*(rPz-rQz)*etha*etha - 4*exponentJJ*exponentLL*(IIz-JJz)*(KKz-LLz)*(rPz-rQz)*(rPz-rQz)*etha*etha + 2*A*(rPz-rQz)*(rPz-rQz)*etha*etha + 2*B*(rPz-rQz)*(rPz-rQz)*etha*etha - 12*(rPz-rQz)*(rPz-rQz)*etha*etha*etha + 4*exponentJJ*(IIz-JJz)*(rPz-rQz)*(rPz-rQz)*(rPz-rQz)*etha*etha*etha - 4*exponentLL*(KKz-LLz)*(rPz-rQz)*(rPz-rQz)*(rPz-rQz)*etha*etha*etha + 4*(rPz-rQz)*(rPz-rQz)*(rPz-rQz)*(rPz-rQz)*etha*etha*etha*etha - 2*exponentKK*(KKz-LLz)*(exponentLL*(KKz-LLz)*(A+etha*(-1 + 2*exponentJJ*(IIz-JJz)*(rPz-rQz) + 2*(rPz-rQz)*(rPz-rQz)*etha)) + etha*(exponentJJ*((IIz-JJz) - 2*(IIz-JJz)*(rPz-rQz)*(rPz-rQz)*etha) - (rPz-rQz)*(A + etha*(-3 + 2*(rPz-rQz)*(rPz-rQz)*etha)))) + 2*exponentII*(IIz-JJz)*(exponentJJ*(IIz-JJz)*(-B + 2*exponentKK*(KKz-LLz)*(exponentLL*(KKz-LLz) - (rPz-rQz)*etha) + etha*(1 + 2*exponentLL*(KKz-LLz)*(rPz-rQz) - 2*(rPz-rQz)*(rPz-rQz)*etha)) + etha*(exponentKK*(KKz-LLz)*(1 + 2*exponentLL*(KKz-LLz)*(rPz-rQz) - 2*(rPz-rQz)*(rPz-rQz)*etha) + exponentLL*(KKz-LLz)*(-1 + 2*(rPz-rQz)*(rPz-rQz)*etha) - (rPz-rQz)*(B + etha*(-3 + 2*(rPz-rQz)*(rPz-rQz)*etha)))))*F;
	  break;
	}

      normIntegral = primNormII*primNormJJ*primNormKK*primNormLL*preIntegral;
      integralValues_d[global1] = coefficientsII*coefficientsJJ*coefficientsKK*coefficientsLL*normIntegral;
    }
}

extern "C" void cuda_int_intraspecies_(int *numberOfContractions,
                                       int *maxNumCartesianOrbital,
				       int *primNormalizationSize,
                                       int *contractionId,
                                       int *contractionLength,
                                       int *contractionAngularMoment,
                                       int *contractionNumCartesianOrbital,
                                       int *contractionOwner,
                                       double *contractionOrigin,
                                       double *contractionOrbitalExponents,
                                       double *contractionCoefficients,
                                       double *contractionContNormalization,
                                       double *contractionPrimNormalization,
				       double *contractionIntegrals,
				       int *contractionIndices)
{
  int N;
  double *integralValues, *integralValues_d;
  int a, b, r, s, u, n;
  int *contLength;
  int contractionsMem, totalPrimitives, unicintegrals, unicintegralsMem, exponentSize;
  int *contIndices, *primIndices, *contCounter;
  double *exponents, *primNormalization, *coefficients, *origin, *contNormalization, *contractedIntegrals, *integralValuesTotal;
  int *angularMoments;
  int *numberOfPPUC, contractionsMemDoub, unicintegralsMemDoub;
  int i,j,k,l,m,p;
  int auxCounter, originSize;

  //Cuda Arrays
  int *contIndices_d, *primIndices_d, *contLength_d, *contCounter_d, *angularMoments_d;
  double *exponents_d, *primNormalization_d, *coefficients_d, *origin_d;

  unicintegrals = ((*numberOfContractions*(*numberOfContractions+1)/2)+1)*(*numberOfContractions*(*numberOfContractions+1)/2)/2;

  //////////////////////////////////////////////////////////////////////
  /// Memory size
  contractionsMem = *numberOfContractions*sizeof(int);
  contractionsMemDoub = *numberOfContractions*sizeof(double);
  unicintegralsMem = unicintegrals*sizeof(int);
  unicintegralsMemDoub = unicintegrals*sizeof(double);
  exponentSize = *primNormalizationSize*sizeof(double);
  originSize = *numberOfContractions*3*sizeof(double);
  //////////////////////////////////////////////////////////////////////

  //////////////////////////////////////////////////////////////////////
  /// Malloc
  //contLength = Contraction size
  contLength = (int *)malloc(contractionsMem);
  // Counter for contractions
  contCounter = (int *)malloc(contractionsMem);
  //numberOfPPC = Number of Primitives per Unic Integral Contraction
  numberOfPPUC = (int *)malloc(unicintegralsMem);
  //Unic Integral Contraction Indices
  contIndices = (int *)malloc(4*unicintegralsMem); 
  //Exponents of contractions
  exponents = (double *)malloc(exponentSize);
  //Primitive normalization constants
  primNormalization = (double *)malloc(exponentSize);
  //Coefficients of contractions
  coefficients = (double *)malloc(exponentSize);
  // Origins of contractions
  origin = (double *)malloc(originSize);
  // Contracted Integrals
  contractedIntegrals = (double *)malloc(unicintegralsMemDoub);
  // Normalization constants of contractions
  contNormalization = (double *)malloc(contractionsMemDoub);
  // Angular moments of contractions
  angularMoments = (int *)malloc(contractionsMem);
  //////////////////////////////////////////////////////////////////////

  auxCounter = 0;
  for(i=0; i<*numberOfContractions;i++)
    {
      contNormalization[i] = *(contractionContNormalization+i);
      angularMoments[i] = *(contractionAngularMoment+i);
      // printf("Angular moments: %d\n", angularMoments[i]);
      for(j=0; j<3; j++)
	{
	  origin[j+i*3] = *(contractionOrigin+(j+i*3));
	  // printf("Origin %f \n",*(contractionOrigin+(j+i*3)));
	}
      contLength[i] = *(contractionLength+i);
      contCounter[i] = auxCounter; 
      // printf("Contraction length: %d %d\n", contLength[i], contCounter[i]);
      // printf("Origins: (%f, %f, %f)\n", origin[i*3], origin[i*3+1], origin[i*3+2]);
      auxCounter += contLength[i];
    }

  // printf("Exponents, coefficients and Primitive Normalization constants:\n");
  for(i=0; i<*primNormalizationSize;i++)
      {
	exponents[i] = *(contractionOrbitalExponents+i);
	primNormalization[i] = *(contractionPrimNormalization+i);
	coefficients[i] = *(contractionCoefficients+i);
	// printf(" (%d) %f %f %f\n", i, exponents[i], coefficients[i], primNormalization[i]);
      }

  m=0;
  totalPrimitives = 0;
  printf("NUmber of Contractions Cudint: %d\n", *numberOfContractions);
  for( a = 1;  a<=*numberOfContractions; a++)
    {
      n = a;
      for( b = a; b<=*numberOfContractions;b++)
  	{
          u = b;
          for( r = n ;r <=*numberOfContractions;r++)
  	    {
  	      for( s = u; s<=*numberOfContractions; s++)
  		{
		  contIndices[m*4] = a;
		  contIndices[m*4+1] = b;
		  contIndices[m*4+2] = r;
		  contIndices[m*4+3] = s;
		  numberOfPPUC[m] = contLength[a-1]*contLength[b-1]*contLength[r-1]*contLength[s-1];
		  totalPrimitives += numberOfPPUC[m];
		  printf("Contraction C (%d): (%d,%d|%d,%d) %d\n", m, a, b, r, s, numberOfPPUC[m] );
		  m++;
  		}
  	      u = r+1;
  	    }
  	}
    }

  m=0;
  p=0;
  primIndices = (int *)malloc(totalPrimitives*5*sizeof(int));
  for( a = 1;  a<=*numberOfContractions; a++)
    {
      n = a;
      for( b = a; b<=*numberOfContractions;b++)
  	{
          u = b;
          for( r = n ;r <=*numberOfContractions;r++)
  	    {
  	      for( s = u; s<=*numberOfContractions; s++)
  		{
		  for(i=1;i<=contLength[a-1];i++)
		    for(j=1;j<=contLength[b-1];j++)
		      for(k=1;k<=contLength[r-1];k++)
			for(l=1;l<=contLength[s-1];l++)
			  {
			    primIndices[5*p] = m;
			    primIndices[5*p+1] = i;
			    primIndices[5*p+2] = j;
			    primIndices[5*p+3] = k;
			    primIndices[5*p+4] = l;
			    // printf("%d, %d, %d, %d\n",i,j,k,l);
			    // printf("Primitives %d, %d, %d, %d, %d\n", primIndices[5*p],
			    	   // primIndices[5*p+1],
			    	   // primIndices[5*p+2],
			    	   // primIndices[5*p+3],
			    	   // primIndices[5*p+4]);
			    p++;
			  }
		  m++;
  		}
  	      u = r+1;
  	    }
  	}
    }


  // printf("Total Primitive: %d\n", totalPrimitives);

  N=totalPrimitives;	  
  integralValuesTotal = (double *)malloc(N*sizeof(double));
  ////////////////////////////////////////////////////////////////////                                                                                                                                                                        /// Total threads in GPUs
  // printf("     *** GPU Especifications ***\n");
  int gpu, count;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&count);
  int totalThreads=0;
  for (gpu = 0; gpu < count; gpu++) {
    hipGetDeviceProperties(&prop,gpu);
    totalThreads+=prop.multiProcessorCount*prop.maxThreadsPerMultiProcessor;
  }
  ////////////////////////////////////////////////////////////////////   
  int numberOfBlocks = totalThreads/numberOfThreads;
  dim3 blockSize(16,16,1);
  dim3 gridSize(numberOfBlocks,1,1);

  ////////////////////////////////////////////////////////////////////////////
  /// CUDA Malloc
  hipMalloc((void **)&primIndices_d, totalPrimitives*5*sizeof(int));
  hipMalloc((void **)&contIndices_d, 4*unicintegralsMem);
  hipMalloc((void **)&exponents_d, exponentSize);
  hipMalloc((void **)&primNormalization_d, exponentSize);
  hipMalloc((void **)&coefficients_d, exponentSize);
  hipMalloc((void **)&contCounter_d, contractionsMem);
  hipMalloc((void **)&angularMoments_d, contractionsMem);
  hipMalloc((void **)&contLength_d, contractionsMem);
  hipMalloc((void **)&origin_d, originSize);
  ///////////////////////////////////////////////////////////////////////////

  ///////////////////////////////////////////////////////////////////////////
  ///CUDA copy
  hipMemcpy(primIndices_d, primIndices, totalPrimitives*5*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(contIndices_d, contIndices, 4*unicintegralsMem, hipMemcpyHostToDevice);
  hipMemcpy(exponents_d, exponents, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(primNormalization_d, primNormalization, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(coefficients_d, coefficients, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(contCounter_d, contCounter, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(angularMoments_d, angularMoments, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(contLength_d, contLength, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(origin_d, origin, originSize, hipMemcpyHostToDevice);
  //////////////////////////////////////////////////////////////////////////

  ////////////////////////////////////////////////////////////////////////
  ///Number of Calls to kernel

  int numberCallkernel = 0;
  
  i=0;
  int kernelIter = 0;
  int control2=0;
  while(control2<=totalPrimitives-1)
    {
      int control = 0;
      kernelIter = control2;
      while(control+numberOfPPUC[i]<=totalThreads && i < unicintegrals)
	{
	  control += numberOfPPUC[i];
          control2 += numberOfPPUC[i];
	  i++;
	  // printf("Control: %d %d\n",i, control);
	}
      numberCallkernel++;
      integralValues = (double *)malloc(control*sizeof(double));
      hipMalloc((void **)&integralValues_d, control*sizeof(double));

      // printf("Control2: %d %d\n", numberCallkernel, control2);

      //      printf("Kernel Call Number: %d\n", numberCallkernel );
      intssss<<<gridSize,blockSize>>>(N, primIndices_d, contIndices_d, exponents_d, primNormalization_d, coefficients_d, contCounter_d, contLength_d, origin_d, angularMoments_d, integralValues_d, control, kernelIter);

      hipMemcpy(integralValues, integralValues_d, control*sizeof(double),hipMemcpyDeviceToHost);
      m=0;

      for(j=kernelIter;j<control2;j++)
	{
	  integralValuesTotal[j] = integralValues[j-kernelIter];    
	  // if(numberCallkernel==3)
	  //    printf("Integral post Kernel: %d, %d -> %f\n", j, j-kernelIter, integralValuesTotal[j]);
	}

      hipFree(integralValues_d);
      free(integralValues);
    }

  printf("Unic Integrals:%d\n", unicintegrals);
      for(i=0; i<unicintegrals;i++)
	{
	  contractedIntegrals[i] = 0.0;
	  a = contIndices[i*4];
	  b = contIndices[i*4+1];
	  r = contIndices[i*4+2];
	  s = contIndices[i*4+3];
	  for(j=0; j<numberOfPPUC[i];j++)
	    {
	      contractedIntegrals[i] += contNormalization[a-1]*contNormalization[b-1]*contNormalization[r-1]*contNormalization[s-1]*integralValuesTotal[m];
	      *(contractionIntegrals+i) = contractedIntegrals[i];
	      *(contractionIndices+(i*4)) = a;
	      *(contractionIndices+(i*4+1)) = b;
	      *(contractionIndices+(i*4+2)) = r;
	      *(contractionIndices+(i*4+3)) = s;
	      m++;
	    }
	  // printf("%d %f %f %f %f\n", i, contNormalization[a],contNormalization[b],contNormalization[r],contNormalization[s]);
	  printf("(%d,%d|%d,%d) = %f \n", a,b,r,s,contractedIntegrals[i]);
	}

  // for(i=0;i<N;i++)
  //   printf("Integral en Host: %d %f\n", i, integralValues[i]);



  hipFree(primIndices_d);
  hipFree(contIndices_d);
  hipFree(exponents_d);
  hipFree(primNormalization_d);
  hipFree(coefficients_d);
  hipFree(contCounter_d);
  hipFree(contLength_d);
  hipFree(origin_d);
  free(integralValuesTotal);
  free(contLength);
  free(contCounter);
  free(numberOfPPUC);
  free(contIndices);
  free(exponents);
  free(primNormalization);
  free(coefficients);
  free(origin);
  free(contractedIntegrals);
  free(contNormalization);

  return;
}
