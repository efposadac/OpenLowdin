#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int numberOfThreads = 256;
const double pi = 3.14159265358979323846;

__global__ void intssss(int N, 
			int *primIndices_d,
			int *contIndices_d,
			double *exponents_d,
			double *primNormalization_d,
			double *coefficients_d,
			int *contCounter_d,
			int *contLength_d,
			double *origin_d,
			double *integralValues_d,
			int control,
			int kernelIter)
{
  int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;
  int blockIndex = blockIdx.x;
  int global1 = threadIndex + blockIndex*blockDim.x*blockDim.y;
  int global = global1 + kernelIter; 
  
  int aa, bb, rr, ss, ii, jj, kk, ll;
  int contractionID;
  double exponentII, exponentJJ, exponentKK, exponentLL;
  double coefficientsII, coefficientsJJ, coefficientsKK, coefficientsLL;
  double primNormII, primNormJJ, primNormKK, primNormLL;
  int exponentIterII, exponentIterJJ, exponentIterKK, exponentIterLL;
  double IIx, IIy, IIz, JJx, JJy, JJz, KKx, KKy, KKz, LLx, LLy, LLz;
  double preIntegral, normIntegral; 

  double A, B, C, D, KIJ, KKL, rPx, rPy, rPz, rQx, rQy, rQz, rPQ, rIJ, rKL, tFunc, tFuncsqrt, F, prefact;

  if(global1< control)
    {
      // ID of unic integrals
      contractionID = primIndices_d[global*5];

      // Contraction Indices
      aa = contIndices_d[contractionID*4];
      bb = contIndices_d[contractionID*4+1];
      rr = contIndices_d[contractionID*4+2];
      ss = contIndices_d[contractionID*4+3];
      // Primitive indices
      ii = primIndices_d[global*5+1];
      jj = primIndices_d[global*5+2];
      kk = primIndices_d[global*5+3];
      ll = primIndices_d[global*5+4];
      
      exponentIterII = contCounter_d[aa-1] + ii - 1;
      exponentIterJJ = contCounter_d[bb-1] + jj - 1;
      exponentIterKK = contCounter_d[rr-1] + kk - 1;
      exponentIterLL = contCounter_d[ss-1] + ll - 1;

      exponentII = exponents_d[exponentIterII];
      exponentJJ = exponents_d[exponentIterJJ];
      exponentKK = exponents_d[exponentIterKK];
      exponentLL = exponents_d[exponentIterLL];

      coefficientsII = coefficients_d[exponentIterII];
      coefficientsJJ = coefficients_d[exponentIterJJ];
      coefficientsKK = coefficients_d[exponentIterKK];
      coefficientsLL = coefficients_d[exponentIterLL];

      primNormII = primNormalization_d[exponentIterII];
      primNormJJ = primNormalization_d[exponentIterJJ];
      primNormKK = primNormalization_d[exponentIterKK];
      primNormLL = primNormalization_d[exponentIterLL];

      IIx = origin_d[(aa*3)-3];
      IIy = origin_d[(aa*3)-2];
      IIz = origin_d[(aa*3)-1];
      JJx = origin_d[(bb*3)-3];
      JJy = origin_d[(bb*3)-2];
      JJz = origin_d[(bb*3)-1];
      KKx = origin_d[(rr*3)-3];
      KKy = origin_d[(rr*3)-2];
      KKz = origin_d[(rr*3)-1];
      LLx = origin_d[(ss*3)-3];
      LLy = origin_d[(ss*3)-2];
      LLz = origin_d[(ss*3)-1];

      
      A = exponentII + exponentJJ;
      B = exponentKK + exponentLL;
      C = exponentII*exponentJJ;
      D = exponentKK*exponentLL;

      rIJ = (IIx-JJx)*(IIx-JJx) + (IIy-JJy)*(IIy-JJy) + (IIz-JJz)*(IIz-JJz);
      rKL = (KKx-LLx)*(KKx-LLx) + (KKy-LLy)*(KKy-LLy) + (KKz-LLz)*(KKz-LLz);

      KIJ = exp(-(C/A)*rIJ);
      KKL = exp(-(D/B)*rKL);

      prefact = (pi*pi*pi)/(A*B*(sqrt(A+B)));

      rPx =(exponentII*IIx+exponentJJ*JJx)/A;
      rPy =(exponentII*IIy+exponentJJ*JJy)/A;
      rPz =(exponentII*IIz+exponentJJ*JJz)/A;
      rQx = (exponentKK*KKx+exponentLL*LLx)/B;
      rQy = (exponentKK*KKy+exponentLL*LLy)/B;
      rQz = (exponentKK*KKz+exponentLL*LLz)/B;
      
      rPQ = (rPx-rQx)*(rPx-rQx) + (rPy-rQy)*(rPy-rQy) + (rPz-rQz)*(rPz-rQz);

      tFunc = (A*B)*rPQ/(A+B);

      tFuncsqrt = sqrt(tFunc);

      if(tFunc == 0.0)
	F = 2/(sqrt(pi));
      else
	F = erf(tFuncsqrt)/tFuncsqrt;
      
      preIntegral = prefact*KIJ*KKL*F;
      normIntegral = primNormII*primNormJJ*primNormKK*primNormLL*preIntegral;
      //integralValues_d[global1] = (double)global;
      integralValues_d[global1] = coefficientsII*coefficientsJJ*coefficientsKK*coefficientsLL*normIntegral;

      // printf("Contraction (%d): (%d,%d|%d,%d)\n Primitive (%d): (%d,%d|%d,%d) [%f, %f, %f | %f, %f, %f | %f, %f, %f | %f, %f, %f]\n",
      // 	     contractionID, aa, bb, rr, ss,
      // 	     global, ii, jj, kk, ll, 
      // 	     IIx, IIy, IIz, JJx, JJy, JJz, KKx, KKy, KKz, LLx, LLy, LLz);
    }
    
}

extern "C" void cuda_int_intraspecies_(int *numberOfContractions,
                                       int *maxNumCartesianOrbital,
				       int *primNormalizationSize,
                                       int *contractionId,
                                       int *contractionLength,
                                       int *contractionAngularMoment,
                                       int *contractionNumCartesianOrbital,
                                       int *contractionOwner,
                                       double *contractionOrigin,
                                       double *contractionOrbitalExponents,
                                       double *contractionCoefficients,
                                       double *contractionContNormalization,
                                       double *contractionPrimNormalization,
				       double *contractionIntegrals,
				       int *contractionIndices)
{
  int N;
  double *integralValues, *integralValues_d;
  int a, b, r, s, u, n;
  int *contLength;
  int contractionsMem, totalPrimitives, unicintegrals, unicintegralsMem, exponentSize;
  int *contIndices, *primIndices, *contCounter;
  double *exponents, *primNormalization, *coefficients, *origin, *contNormalization, *contractedIntegrals, *integralValuesTotal;
  int *numberOfPPUC, contractionsMemDoub, unicintegralsMemDoub;
  int i,j,k,l,m,p;
  int auxCounter, originSize;

  //Cuda Arrays
  int *contIndices_d, *primIndices_d, *contLength_d, *contCounter_d;
  double *exponents_d, *primNormalization_d, *coefficients_d, *origin_d;

  unicintegrals = ((*numberOfContractions*(*numberOfContractions+1)/2)+1)*(*numberOfContractions*(*numberOfContractions+1)/2)/2;

  //////////////////////////////////////////////////////////////////////
  /// Memory size
  contractionsMem = *numberOfContractions*sizeof(int);
  contractionsMemDoub = *numberOfContractions*sizeof(double);
  unicintegralsMem = unicintegrals*sizeof(int);
  unicintegralsMemDoub = unicintegrals*sizeof(double);
  exponentSize = *primNormalizationSize*sizeof(double);
  originSize = *numberOfContractions*3*sizeof(double);
  //////////////////////////////////////////////////////////////////////

  //////////////////////////////////////////////////////////////////////
  /// Malloc
  //contLength = Contraction size
  contLength = (int *)malloc(contractionsMem);
  // Counter for contractions
  contCounter = (int *)malloc(contractionsMem);
  //numberOfPPC = Number of Primitives per Unic Integral Contraction
  numberOfPPUC = (int *)malloc(unicintegralsMem);
  //Unic Integral Contraction Indices
  contIndices = (int *)malloc(4*unicintegralsMem); 
  //Exponents of contractions
  exponents = (double *)malloc(exponentSize);
  //Primitive normalization constants
  primNormalization = (double *)malloc(exponentSize);
  //Coefficients of contractions
  coefficients = (double *)malloc(exponentSize);
  // Origins of contractions
  origin = (double *)malloc(originSize);
  // Contracted Integrals
  contractedIntegrals = (double *)malloc(unicintegralsMemDoub);
  // Normalization constants of contractions
  contNormalization = (double *)malloc(contractionsMemDoub);
  //////////////////////////////////////////////////////////////////////

  auxCounter = 0;
  for(i=0; i<*numberOfContractions;i++)
    {
      contNormalization[i] = *(contractionContNormalization+i);
      for(j=0; j<3; j++)
	{
	  origin[j+i*3] = *(contractionOrigin+(j+i*3));
	  // printf("Origin %f \n",*(contractionOrigin+(j+i*3)));
	}
      contLength[i] = *(contractionLength+i);
      contCounter[i] = auxCounter; 
      // printf("Contraction length: %d %d\n", contLength[i], contCounter[i]);
      // printf("Origins: (%f, %f, %f)\n", origin[i*3], origin[i*3+1], origin[i*3+2]);
      auxCounter += contLength[i];
    }

  // printf("Exponents, coefficients and Primitive Normalization constants:\n");
  for(i=0; i<*primNormalizationSize;i++)
      {
	exponents[i] = *(contractionOrbitalExponents+i);
	primNormalization[i] = *(contractionPrimNormalization+i);
	coefficients[i] = *(contractionCoefficients+i);
	// printf(" (%d) %f %f %f\n", i, exponents[i], coefficients[i], primNormalization[i]);
      }

  m=0;
  totalPrimitives = 0;
  for( a = 1;  a<=*numberOfContractions; a++)
    {
      n = a;
      for( b = a; b<=*numberOfContractions;b++)
  	{
          u = b;
          for( r = n ;r <=*numberOfContractions;r++)
  	    {
  	      for( s = u; s<=*numberOfContractions; s++)
  		{
		  contIndices[m*4] = a;
		  contIndices[m*4+1] = b;
		  contIndices[m*4+2] = r;
		  contIndices[m*4+3] = s;
		  numberOfPPUC[m] = contLength[a-1]*contLength[b-1]*contLength[r-1]*contLength[s-1];
		  totalPrimitives += numberOfPPUC[m];
		  // printf("Contraction C (%d): (%d,%d|%d,%d) %d\n", m, a, b, r, s, numberOfPPUC[m] );
		  m++;
  		}
  	      u = r+1;
  	    }
  	}
    }

  m=0;
  p=0;
  primIndices = (int *)malloc(totalPrimitives*5*sizeof(int));
  for( a = 1;  a<=*numberOfContractions; a++)
    {
      n = a;
      for( b = a; b<=*numberOfContractions;b++)
  	{
          u = b;
          for( r = n ;r <=*numberOfContractions;r++)
  	    {
  	      for( s = u; s<=*numberOfContractions; s++)
  		{
		  for(i=1;i<=contLength[a-1];i++)
		    for(j=1;j<=contLength[b-1];j++)
		      for(k=1;k<=contLength[r-1];k++)
			for(l=1;l<=contLength[s-1];l++)
			  {
			    primIndices[5*p] = m;
			    primIndices[5*p+1] = i;
			    primIndices[5*p+2] = j;
			    primIndices[5*p+3] = k;
			    primIndices[5*p+4] = l;
			    // printf("%d, %d, %d, %d\n",i,j,k,l);
			    // printf("Primitives %d, %d, %d, %d, %d\n", primIndices[5*p],
			    	   // primIndices[5*p+1],
			    	   // primIndices[5*p+2],
			    	   // primIndices[5*p+3],
			    	   // primIndices[5*p+4]);
			    p++;
			  }
		  m++;
  		}
  	      u = r+1;
  	    }
  	}
    }


  printf("Total Primitive: %d\n", totalPrimitives);

  N=totalPrimitives;	  
  integralValuesTotal = (double *)malloc(N*sizeof(double));
  ////////////////////////////////////////////////////////////////////                                                                                                                                                                        /// Total threads in GPUs
  // printf("     *** GPU Especifications ***\n");
  int gpu, count;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&count);
  int totalThreads=0;
  for (gpu = 0; gpu < count; gpu++) {
    hipGetDeviceProperties(&prop,gpu);
    totalThreads+=prop.multiProcessorCount*prop.maxThreadsPerMultiProcessor;
  }
  ////////////////////////////////////////////////////////////////////   
  int numberOfBlocks = totalThreads/numberOfThreads;
  dim3 blockSize(16,16,1);
  dim3 gridSize(numberOfBlocks,1,1);

  ////////////////////////////////////////////////////////////////////////////
  /// CUDA Malloc
  hipMalloc((void **)&primIndices_d, totalPrimitives*5*sizeof(int));
  hipMalloc((void **)&contIndices_d, 4*unicintegralsMem);
  hipMalloc((void **)&exponents_d, exponentSize);
  hipMalloc((void **)&primNormalization_d, exponentSize);
  hipMalloc((void **)&coefficients_d, exponentSize);
  hipMalloc((void **)&contCounter_d, contractionsMem);
  hipMalloc((void **)&contLength_d, contractionsMem);
  hipMalloc((void **)&origin_d, originSize);
  ///////////////////////////////////////////////////////////////////////////

  ///////////////////////////////////////////////////////////////////////////
  ///CUDA copy
  hipMemcpy(primIndices_d, primIndices, totalPrimitives*5*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(contIndices_d, contIndices, 4*unicintegralsMem, hipMemcpyHostToDevice);
  hipMemcpy(exponents_d, exponents, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(primNormalization_d, primNormalization, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(coefficients_d, coefficients, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(contCounter_d, contCounter, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(contLength_d, contLength, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(origin_d, origin, originSize, hipMemcpyHostToDevice);
  //////////////////////////////////////////////////////////////////////////

  ////////////////////////////////////////////////////////////////////////
  ///Number of Calls to kernel

  int numberCallkernel = 0;
  
  i=0;
  int kernelIter = 0;
  int control2=0;
  while(control2<=totalPrimitives-1)
    {
      int control = 0;
      kernelIter = control2;
      while(control+numberOfPPUC[i]<=totalThreads && i < unicintegrals)
	{
	  control += numberOfPPUC[i];
          control2 += numberOfPPUC[i];
	  i++;
	  // printf("Control: %d %d\n",i, control);
	}
      numberCallkernel++;
      integralValues = (double *)malloc(control*sizeof(double));
      hipMalloc((void **)&integralValues_d, control*sizeof(double));

      // printf("Control2: %d %d\n", numberCallkernel, control2);

      //      printf("Kernel Call Number: %d\n", numberCallkernel );
      intssss<<<gridSize,blockSize>>>(N, primIndices_d, contIndices_d, exponents_d, primNormalization_d, coefficients_d, contCounter_d, contLength_d, origin_d, integralValues_d, control, kernelIter);

      hipMemcpy(integralValues, integralValues_d, control*sizeof(double),hipMemcpyDeviceToHost);
      m=0;

      for(j=kernelIter;j<control2;j++)
	{
	  integralValuesTotal[j] = integralValues[j-kernelIter];    
	  // if(numberCallkernel==3)
	  //    printf("Integral post Kernel: %d, %d -> %f\n", j, j-kernelIter, integralValuesTotal[j]);
	}

      hipFree(integralValues_d);
      free(integralValues);
    }

      // printf("Contracted Integrals:\n");
      for(i=0; i<unicintegrals;i++)
	{
	  contractedIntegrals[i] = 0.0;
	  a = contIndices[i*4];
	  b = contIndices[i*4+1];
	  r = contIndices[i*4+2];
	  s = contIndices[i*4+3];
	  for(j=0; j<numberOfPPUC[i];j++)
	    {
	      contractedIntegrals[i] += contNormalization[a-1]*contNormalization[b-1]*contNormalization[r-1]*contNormalization[s-1]*integralValuesTotal[m];
	      *(contractionIntegrals+i) = contractedIntegrals[i];
	      *(contractionIndices+(i*4)) = a;
	      *(contractionIndices+(i*4+1)) = b;
	      *(contractionIndices+(i*4+2)) = r;
	      *(contractionIndices+(i*4+3)) = s;
	      m++;
	    }
	  // printf("%d %f %f %f %f\n", i, contNormalization[a],contNormalization[b],contNormalization[r],contNormalization[s]);
	  // printf("(%d,%d|%d,%d) = %f \n", a,b,r,s,contractedIntegrals[i]);
	}

  // for(i=0;i<N;i++)
  //   printf("Integral en Host: %d %f\n", i, integralValues[i]);



  hipFree(primIndices_d);
  hipFree(contIndices_d);
  hipFree(exponents_d);
  hipFree(primNormalization_d);
  hipFree(coefficients_d);
  hipFree(contCounter_d);
  hipFree(contLength_d);
  hipFree(origin_d);
  free(integralValuesTotal);
  free(contLength);
  free(contCounter);
  free(numberOfPPUC);
  free(contIndices);
  free(exponents);
  free(primNormalization);
  free(coefficients);
  free(origin);
  free(contractedIntegrals);
  free(contNormalization);

  return;
}
