#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int numberOfThreads = 64;

__global__ void intssss(int N, int *primIndices_d, int *contIndices_d, double *integralValues_d)
{
  int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;
  int blockIndex = blockIdx.x;
  int global = threadIndex + blockIndex*blockDim.x*blockDim.y;
  
  // if(global<N)
    
}

extern "C" void cuda_int_intraspecies_(int *numberOfContractions,
                                       int *maxLength,
                                       int *maxNumCartesianOrbital,
				       int *primNormalizationSize,
                                       int *contractionId,
                                       int *contractionLength,
                                       int *contractionAngularMoment,
                                       int *contractionNumCartesianOrbital,
                                       int *contractionOwner,
                                       double *contractionOrigin,
                                       double *contractionOrbitalExponents,
                                       double *contractionCoefficients,
                                       double *contractionContNormalization,
                                       double *contractionPrimNormalization)
{
  int N;
  double *integralValues, *integralValues_d;
  int a, b, r, s, u, n;
  int *contLength;
  int contractionsMem, totalPrimitives, unicintegrals, unicintegralsMem, exponentSize;
  int *contIndices, *primIndices;
  double *exponents;
  int *numberOfPPUC;
  int i,j,k,l,m,p;

  //Cuda Arrays
  int *contIndices_d, *primIndices_d;

  unicintegrals = ((*numberOfContractions*(*numberOfContractions+1)/2)+1)*(*numberOfContractions*(*numberOfContractions+1)/2)/2;

  contractionsMem = *numberOfContractions*sizeof(int);
  unicintegralsMem = unicintegrals*sizeof(int);

  //////////////////////////////////////////////////////////////////////
  /// Malloc
  //contLength = Contraction size
  contLength = (int *)malloc(contractionsMem);
  //numberOfPPC = Number of Primitives per Unic Integral Contraction
  numberOfPPUC = (int *)malloc(unicintegralsMem);
  //Unic Integral Contraction Indices
  contIndices = (int *)malloc(4*unicintegralsMem); 
  //////////////////////////////////////////////////////////////////////

  exponentSize = 0;
  for(i=0; i<*numberOfContractions;i++)
    {
      contLength[i] = *(contractionLength+i);
      exponentSize += contLength[i];
      printf("Contraction length: %d %d\n", contLength[i], exponentSize);
    }

  int counter=0;
  exponents = (double *)malloc(exponentSize*sizeof(double));
  m=0;
  for(i=0; i<*numberOfContractions;i++){
    for(j=0; j<*maxLength;j++)
      {
	printf(" (%d, %d) %d, %d %f",i,j, *maxLength, *numberOfContractions, *(contractionOrbitalExponents+(j+i*(*numberOfContractions))));
	m++;	     }
	  printf("\n");
      }

	  m=0;

  for(i=0; i<*numberOfContractions;i++)
    {
      for(j=counter;j<(counter+contLength[i]);j++)
	{
	  exponents[m] = *(contractionOrbitalExponents+(j+i*(*numberOfContractions)));
	  printf("Exponent: %f %d, %d\n", exponents[m], m, j);
	  m++;
	}
      counter += *maxLength;
    }

  m=0;
  totalPrimitives = 0;
  for( a = 1;  a<=*numberOfContractions; a++)
    {
      n = a;
      for( b = a; b<=*numberOfContractions;b++)
  	{
          u = b;
          for( r = n ;r <=*numberOfContractions;r++)
  	    {
  	      for( s = u; s<=*numberOfContractions; s++)
  		{
		  contIndices[m*4] = a;
		  contIndices[m*4+1] = b;
		  contIndices[m*4+2] = r;
		  contIndices[m*4+3] = s;
		  numberOfPPUC[m] = contLength[a-1]*contLength[b-1]*contLength[r-1]*contLength[s-1];
		  totalPrimitives += numberOfPPUC[m];
		  printf("Primitives per contraction: %d, %d, %d, %d, %d\n", numberOfPPUC[m], a, b, r, s );
		  m++;
  		}
  	      u = r+1;
  	    }
  	}
    }

  m=0;
  p=0;
  primIndices = (int *)malloc(totalPrimitives*5*sizeof(int));
  for( a = 1;  a<=*numberOfContractions; a++)
    {
      n = a;
      for( b = a; b<=*numberOfContractions;b++)
  	{
          u = b;
          for( r = n ;r <=*numberOfContractions;r++)
  	    {
  	      for( s = u; s<=*numberOfContractions; s++)
  		{
		  for(i=1;i<=contLength[a-1];i++)
		    for(j=1;j<=contLength[b-1];j++)
		      for(k=1;k<=contLength[r-1];k++)
			for(l=1;l<=contLength[s-1];l++)
			  {
			    primIndices[5*p] = m;
			    primIndices[5*p+1] = i;
			    primIndices[5*p+2] = j;
			    primIndices[5*p+3] = k;
			    primIndices[5*p+4] = l;
			    // printf("%d, %d, %d, %d\n",i,j,k,l);
			    printf("Primitives %d, %d, %d, %d, %d\n", primIndices[5*p],
			    	   primIndices[5*p+1],
			    	   primIndices[5*p+2],
			    	   primIndices[5*p+3],
			    	   primIndices[5*p+4]);
			    p++;
			  }
		  m++;
  		}
  	      u = r+1;
  	    }
  	}
    }


  // printf("Total Primitive: %d\n", totalPrimitives);

  N=totalPrimitives;	  
  integralValues = (double *)malloc(totalPrimitives*sizeof(double));

  ////////////////////////////////////////////////////////////////////////////
  /// CUDA Malloc
  hipMalloc((void **)&integralValues_d, totalPrimitives*sizeof(double));
  hipMalloc((void **)&primIndices_d, totalPrimitives*5*sizeof(int));
  hipMalloc((void **)&contIndices_d, 4*unicintegralsMem);
  ///////////////////////////////////////////////////////////////////////////

  ///////////////////////////////////////////////////////////////////////////
  ///CUDA copy
  hipMemcpy(primIndices_d, primIndices, totalPrimitives*5*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(contIndices_d, contIndices, 4*unicintegralsMem, hipMemcpyHostToDevice);
  //////////////////////////////////////////////////////////////////////////

  dim3 blockSize(8,8,1);
  dim3 gridSize(360,1,1);

  intssss<<<gridSize,blockSize>>>(N, primIndices_d, contIndices_d, integralValues);

  hipMemcpy(integralValues, integralValues_d, totalPrimitives*sizeof(double),hipMemcpyDeviceToHost);

  hipFree(integralValues_d);
  free(integralValues);
}
