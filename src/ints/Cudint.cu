#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int numberOfThreads = 256;
const double pi = 3.14159265358979323846;

__device__ float kroneckerDelta(int i, int j)
{
  double delta;

  delta = 1.0;
  if(i != j)
    delta = 0.0;

  return delta;
}

__device__ float errorFunction(int order, double tFunc)
{

  double tFuncsqrt;
  double errorF; 

  tFuncsqrt = sqrt(tFunc);

  if(round(tFunc) == 0.0)
    errorF = 1.0/(2*order + 1);
  else
    {
      switch(order)
	{
	case 0:
	  errorF = 0.5*erf(tFuncsqrt)*sqrt(pi/tFunc);
	  break;
	case 1:
	  errorF = 0.25*(-2*tFuncsqrt*exp(-tFunc) + sqrt(pi)*erf(tFuncsqrt))/(tFuncsqrt*tFunc);
	  break;
	case 2:
	  errorF = -0.125*(exp(-tFunc)*(6*tFuncsqrt + 4*tFuncsqrt*tFunc) - 3*sqrt(pi)*erf(tFuncsqrt))/(tFuncsqrt*tFunc*tFunc);
	  break;
	case 3:
	  errorF = -0.0625*(exp(-tFunc)*(30*tFuncsqrt + 20*tFunc*tFuncsqrt + 8*tFunc*tFunc*tFuncsqrt) - 15*sqrt(pi)*erf(tFuncsqrt))/(tFuncsqrt*tFunc*tFunc*tFunc);
	  break;
	case 4:
	  errorF = -0.03125*(exp(-tFunc)*(16*tFuncsqrt*tFunc*tFunc*tFunc + 56*tFunc*tFunc*tFuncsqrt + 140*tFunc*tFuncsqrt + 210*tFuncsqrt) - (105*sqrt(pi)*erf(tFuncsqrt)))/(tFuncsqrt*tFunc*tFunc*tFunc*tFunc);
	  break;
	}
    }

  return errorF;
}

__global__ void analyticInts(int N, 
			int *primIndices_d,
			int *contIndices_d,
			double *exponents_d,
			double *primNormalization_d,
			double *coefficients_d,
			int *contCounter_d,
			int *contLength_d,
			double *origin_d,
			int *angularMoments_d,
			double *integralValues_d,
			int control,
			int kernelIter)
{
  int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;
  int blockIndex = blockIdx.x;
  int global1 = threadIndex + blockIndex*blockDim.x*blockDim.y;
  int global = global1 + kernelIter; 
  
  int aa, bb, rr, ss, ii, jj, kk, ll;
  int contractionID;
  double exponentII, exponentJJ, exponentKK, exponentLL;
  double coefficientsII, coefficientsJJ, coefficientsKK, coefficientsLL;
  double primNormII, primNormJJ, primNormKK, primNormLL;
  int exponentIterII, exponentIterJJ, exponentIterKK, exponentIterLL;
  double IIx, IIy, IIz, JJx, JJy, JJz, KKx, KKy, KKz, LLx, LLy, LLz;
  double preIntegral, normIntegral; 
  double etha;
  int lAA, lBB, lRR, lSS; // Angular moments of contractions
  int integralCase;

  double A, B, C, D, KIJ, KKL, rPx, rPy, rPz, rQx, rQy, rQz, rPQ, rIJ, rKL, tFunc, prefact;
  double FA, FB, FC, FD, FE;
  double rPQx, rPQy, rPQz;
  int alpha, beta, kappa, lambda, selectCart;
  double dij;

  if(global1< control)
    {
      // ID of unic integrals
      contractionID = primIndices_d[global*9];

      // Contraction Indices
      aa = contIndices_d[contractionID*4];
      bb = contIndices_d[contractionID*4+1];
      rr = contIndices_d[contractionID*4+2];
      ss = contIndices_d[contractionID*4+3];
      
      // Primitive indices
      ii = primIndices_d[global*9+1];
      jj = primIndices_d[global*9+2];
      kk = primIndices_d[global*9+3];
      ll = primIndices_d[global*9+4];
      
      // Label of cartesian
      alpha = primIndices_d[global*9+5];
      beta = primIndices_d[global*9+6];
      kappa = primIndices_d[global*9+7];
      lambda = primIndices_d[global*9+8];

      lAA = angularMoments_d[aa-1];
      lBB = angularMoments_d[bb-1];
      lRR = angularMoments_d[rr-1];
      lSS = angularMoments_d[ss-1];
      
      exponentIterII = contCounter_d[aa-1] + ii - 1;
      exponentIterJJ = contCounter_d[bb-1] + jj - 1;
      exponentIterKK = contCounter_d[rr-1] + kk - 1;
      exponentIterLL = contCounter_d[ss-1] + ll - 1;

      exponentII = exponents_d[exponentIterII];
      exponentJJ = exponents_d[exponentIterJJ];
      exponentKK = exponents_d[exponentIterKK];
      exponentLL = exponents_d[exponentIterLL];

      coefficientsII = coefficients_d[exponentIterII];
      coefficientsJJ = coefficients_d[exponentIterJJ];
      coefficientsKK = coefficients_d[exponentIterKK];
      coefficientsLL = coefficients_d[exponentIterLL];

      primNormII = primNormalization_d[exponentIterII];
      primNormJJ = primNormalization_d[exponentIterJJ];
      primNormKK = primNormalization_d[exponentIterKK];
      primNormLL = primNormalization_d[exponentIterLL];

      IIx = origin_d[(aa*3)-3];
      IIy = origin_d[(aa*3)-2];
      IIz = origin_d[(aa*3)-1];
      JJx = origin_d[(bb*3)-3];
      JJy = origin_d[(bb*3)-2];
      JJz = origin_d[(bb*3)-1];
      KKx = origin_d[(rr*3)-3];
      KKy = origin_d[(rr*3)-2];
      KKz = origin_d[(rr*3)-1];
      LLx = origin_d[(ss*3)-3];
      LLy = origin_d[(ss*3)-2];
      LLz = origin_d[(ss*3)-1];
    
      A = exponentII + exponentJJ;
      B = exponentKK + exponentLL;
      C = exponentII*exponentJJ;
      D = exponentKK*exponentLL;

      etha = (A*B)/(A+B);

      rIJ = (IIx-JJx)*(IIx-JJx) + (IIy-JJy)*(IIy-JJy) + (IIz-JJz)*(IIz-JJz);
      rKL = (KKx-LLx)*(KKx-LLx) + (KKy-LLy)*(KKy-LLy) + (KKz-LLz)*(KKz-LLz);

      KIJ = exp(-(C/A)*rIJ);
      KKL = exp(-(D/B)*rKL);

      prefact = sqrt(etha/pi)*sqrt(pi/A)*(pi/A)*sqrt(pi/B)*(pi/B)*KIJ*KKL;
      // if(aa==1 && bb==1 && rr==1 && ss==2)
      // 	printf("etha: %f KIJ: %f KKL: %f prefact: %f %f %f %f\n", etha, KIJ, KKL, prefact, D, B, rKL);
      
      rPx =(exponentII*IIx+exponentJJ*JJx)/A;
      rPy =(exponentII*IIy+exponentJJ*JJy)/A;
      rPz =(exponentII*IIz+exponentJJ*JJz)/A;
      rQx = (exponentKK*KKx+exponentLL*LLx)/B;
      rQy = (exponentKK*KKy+exponentLL*LLy)/B;
      rQz = (exponentKK*KKz+exponentLL*LLz)/B;
      
      rPQx = (rPx*A + rQx*B)/(A+B);
      rPQy = (rPy*A + rQy*B)/(A+B);
      rPQz = (rPz*A + rQz*B)/(A+B);

      rPQ = (rPx-rQx)*(rPx-rQx) + (rPy-rQy)*(rPy-rQy) + (rPz-rQz)*(rPz-rQz);

      tFunc = 0.0;
      tFunc = etha*rPQ;

      FA = 0.0;
      FB = 0.0;
      FC = 0.0;
      FD = 0.0;
      FE = 0.0;
      
      integralCase = 64*lAA + 16*lBB + 4*lRR + lSS;

      switch(integralCase)
	{
	case 0: // Integral (s,s|s,s)
	  FA = errorFunction(0, tFunc);
	  preIntegral = 2*FA*prefact;
	  break;
	case 64:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  switch(alpha)
	    {
	    case 1: // Integral (px,s|s,s)
	      preIntegral = 2*(FB*(rPQx-rPx)+FA*(rPx-IIx))*prefact;
	      break;
	    case 2: // Integral (py,s|s,s)
	      preIntegral = 2*(FB*(rPQy-rPy)+FA*(rPy-IIy))*prefact;
	      break;
	    case 3: // Integral (pz,s|s,s)
	      preIntegral = 2*(FB*(rPQz-rPz)+FA*(rPz-IIz))*prefact;
	      break;
	    }
	  break;
	case 68:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  FC = errorFunction(2, tFunc);
	  selectCart = 64*alpha + 4*kappa;
	  dij = kroneckerDelta(alpha, kappa);
	  switch(selectCart)
	    {
	    case 68: // Integral (px,s|px,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQx-KKx)*(FB*(rPQx-rPx) + FA*(rPx-IIx)) + 2*(rPQx-rQx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)));
	      break;
	    case 72: // Integral (px,s|py,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQy-KKy)*(FB*(rPQx-rPx) + FA*(rPx-IIx)) + 2*(rPQy-rQy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)));
	      break;
	    case 76: // Integral (px,s|pz,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQz-KKz)*(FB*(rPQx-rPx) + FA*(rPx-IIx)) + 2*(rPQz-rQz)*(FC*(rPQx-rPx) + FB*(rPx-IIx)));
	      break;
	    case 136: // Integral (py,s|py,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQy-KKy)*(FB*(rPQy-rPy) + FA*(rPy-IIy)) + 2*(rPQy-rQy)*(FC*(rPQy-rPy) + FB*(rPy-IIy)));
	      break;
	    case 140: // Integral (py,s|pz,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQz-KKz)*(FB*(rPQy-rPy) + FA*(rPy-IIy)) + 2*(rPQz-rQz)*(FC*(rPQy-rPy) + FB*(rPy-IIy)));
	      break;
	    case 204: // Integral (pz,s|pz,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQz-KKz)*(FB*(rPQz-rPz) + FA*(rPz-IIz)) + 2*(rPQz-rQz)*(FC*(rPQz-rPz) + FB*(rPz-IIz)));
	      break;
	    }
	  break;
	case 80:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  FC = errorFunction(2, tFunc);
	  selectCart = 64*alpha + 16*beta;
	  dij = kroneckerDelta(alpha, beta);
	  switch(selectCart)
	    {
	    case 80: // Integral (px,px|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) + 2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx));
	      break;
	    case 96: // Integral (px,py|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) + 2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy));
	      break;
	    case 112: // Integral (px,pz|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) + 2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz));
	      break;
	    case 160: // Integral (py,py|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) + 2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy));
	      break;
	    case 176: // Integral (py,pz|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) + 2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz));
	      break;
	    case 240: // Integral (pz,pz|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) + 2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz));
	      break;
	    }
	  break;
	}
      // if(aa == 1 && bb == 1 && rr == 1 && ss == 2)
      // 	{
      // 	  printf("Sin Norm:  %f %f | %f %f\n",
      // 		 preIntegral, prefact, FA, rKL);
      // 	}
      normIntegral = primNormII*primNormJJ*primNormKK*primNormLL*preIntegral;
      integralValues_d[global1] = coefficientsII*coefficientsJJ*coefficientsKK*coefficientsLL*normIntegral;
    }
}

extern "C" void cuda_int_intraspecies_(int *numberOfContractions,
				       int *totalContIntegrals,
				       int *totalPrimitives,
                                       int *maxNumCartesianOrbital,
				       int *primNormalizationSize,
                                       int *contractionId,
                                       int *contractionLength,
                                       int *contractionAngularMoment,
                                       int *contractionNumCartesianOrbital,
                                       int *contractionOwner,
                                       double *contractionOrigin,
                                       double *contractionOrbitalExponents,
                                       double *contractionCoefficients,
                                       double *contractionContNormalization,
                                       double *contractionPrimNormalization,
				       double *contractionIntegrals,
				       int *contractionIndices, 
				       int *primitiveIndices,
				       int *numberOfPPUC,
				       int *labelsOfContractions)
{
  int N;
  double *integralValues, *integralValues_d;
  int a, b, r, s;
  int i,j;
  int m;
  int *contLength;
  int totalPrim;
  int contractionsMem, unicintegrals, unicintegralsMem, exponentSize;
  int *contIndices, *primIndices, *contCounter;
  double *exponents, *primNormalization, *coefficients, *origin, *contNormalization, *contractedIntegrals, *integralValuesTotal;
  int *angularMoments;
  int *numCartesianOrbitals, *labelsForContractions;
  int *auxNumberOfPPUC, contractionsMemDoub, unicintegralsMemDoub;
  int auxCounter, originSize;

  //Cuda Arrays
  int *contIndices_d, *primIndices_d, *contLength_d, *contCounter_d, *angularMoments_d;
  double *exponents_d, *primNormalization_d, *coefficients_d, *origin_d;

  // unicintegrals = ((*numberOfContractions*(*numberOfContractions+1)/2)+1)*(*numberOfContractions*(*numberOfContractions+1)/2)/2;
  unicintegrals = *totalContIntegrals;
  totalPrim = *totalPrimitives;

  //////////////////////////////////////////////////////////////////////
  /// Memory size
  contractionsMem = *numberOfContractions*sizeof(int);
  contractionsMemDoub = *numberOfContractions*sizeof(double);
  unicintegralsMem = unicintegrals*sizeof(int);
  unicintegralsMemDoub = unicintegrals*sizeof(double);
  exponentSize = *primNormalizationSize*sizeof(double);
  originSize = *numberOfContractions*3*sizeof(double);
  //////////////////////////////////////////////////////////////////////

  //////////////////////////////////////////////////////////////////////
  /// Malloc
  //contLength = Contraction size
  contLength = (int *)malloc(contractionsMem);
  // Counter for contractions
  contCounter = (int *)malloc(contractionsMem);
  //numberOfPPC = Number of Primitives per Unic Integral Contraction
  auxNumberOfPPUC = (int *)malloc(3*unicintegralsMem);
  //Unic Integral Contraction Indices
  contIndices = (int *)malloc(4*unicintegralsMem); 
  //Exponents of contractions
  exponents = (double *)malloc(exponentSize);
  //Primitive normalization constants
  primNormalization = (double *)malloc(exponentSize);
  //Coefficients of contractions
  coefficients = (double *)malloc(exponentSize);
  // Origins of contractions
  origin = (double *)malloc(originSize);
  // Contracted Integrals
  contractedIntegrals = (double *)malloc(unicintegralsMemDoub);
  // Normalization constants of contractions
  contNormalization = (double *)malloc(contractionsMemDoub);
  // Angular moments of contractions
  angularMoments = (int *)malloc(contractionsMem);
  // Number of cartesian orbitals
  numCartesianOrbitals = (int *)malloc(contractionsMem);
  // Labels of cartesian orbitals
  labelsForContractions = (int *)malloc(contractionsMem);
  //////////////////////////////////////////////////////////////////////

  auxCounter = 0;
  for(i=0; i<*numberOfContractions;i++)
    {
      contNormalization[i] = *(contractionContNormalization+i);
      angularMoments[i] = *(contractionAngularMoment+i);
      numCartesianOrbitals[i] = *(contractionNumCartesianOrbital+i);
      labelsForContractions[i] = *(labelsOfContractions+i);
      // printf("Angular moments: %d\n", angularMoments[i]);
      for(j=0; j<3; j++)
	{
	  origin[j+i*3] = *(contractionOrigin+(j+i*3));
             printf("Origin from inter %f \n",*(contractionOrigin+(j+i*3)), origin[j+i*3]);
	}
      contLength[i] = *(contractionLength+i);
      contCounter[i] = auxCounter; 
      // printf("Contraction length: %d %d\n", contLength[i], contCounter[i]);
      printf("Origins: (%f, %f, %f)\n", origin[i*3], origin[i*3+1], origin[i*3+2]);
      auxCounter += contLength[i];
    }

  // printf("Exponents, coefficients and Primitive Normalization constants:\n");
  for(i=0; i<*primNormalizationSize;i++)
      {
	exponents[i] = *(contractionOrbitalExponents+i);
	primNormalization[i] = *(contractionPrimNormalization+i);
	coefficients[i] = *(contractionCoefficients+i);
	// printf(" (%d) %f %f %f\n", i, exponents[i], coefficients[i], primNormalization[i]);
      }

  m=0;
  for( i=0; i<unicintegrals; i++ )
    {
      contIndices[i*4] = *(contractionIndices+(i*4));
      contIndices[i*4+1] = *(contractionIndices+(i*4+1));
      contIndices[i*4+2] = *(contractionIndices+(i*4+2));
      contIndices[i*4+3] = *(contractionIndices+(i*4+3));
      auxNumberOfPPUC[i*3] = *(numberOfPPUC+(i*3));
      auxNumberOfPPUC[i*3+1] = *(numberOfPPUC+(i*3+1));
      auxNumberOfPPUC[i*3+2] = *(numberOfPPUC+(i*3+2));
      /* printf("Contraction num: %d (%d,%d|%d,%d)\n", i, contIndices[i*4], contIndices[i*4+1], contIndices[i*4+2], contIndices[i*4+3]); */
    }

  primIndices = (int *)malloc(totalPrim*9*sizeof(int));
  for( i=0; i<totalPrim; i++)
    {
      primIndices[i*9] = *(primitiveIndices+(i*9));
      primIndices[i*9+1] = *(primitiveIndices+(i*9+1));
      primIndices[i*9+2] = *(primitiveIndices+(i*9+2));
      primIndices[i*9+3] = *(primitiveIndices+(i*9+3));
      primIndices[i*9+4] = *(primitiveIndices+(i*9+4));
      primIndices[i*9+5] = *(primitiveIndices+(i*9+5));
      primIndices[i*9+6] = *(primitiveIndices+(i*9+6));
      primIndices[i*9+7] = *(primitiveIndices+(i*9+7));
      primIndices[i*9+8] = *(primitiveIndices+(i*9+8));
    }

  N=totalPrim;	  
  integralValuesTotal = (double *)malloc(N*sizeof(double));
  ////////////////////////////////////////////////////////////////////
  /// Total threads in GPUs
  // printf("     *** GPU Especifications ***\n");
  int gpu, count;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&count);
  int totalThreads=0;
  for (gpu = 0; gpu < count; gpu++) {
    hipGetDeviceProperties(&prop,gpu);
    totalThreads+=prop.multiProcessorCount*prop.maxThreadsPerMultiProcessor;
  }
  ////////////////////////////////////////////////////////////////////   
  int numberOfBlocks = totalThreads/numberOfThreads;
  dim3 blockSize(16,16,1);
  dim3 gridSize(numberOfBlocks,1,1);

  ////////////////////////////////////////////////////////////////////////////
  /// CUDA Malloc
  hipMalloc((void **)&primIndices_d, totalPrim*9*sizeof(int));
  hipMalloc((void **)&contIndices_d, 4*unicintegralsMem);
  hipMalloc((void **)&exponents_d, exponentSize);
  hipMalloc((void **)&primNormalization_d, exponentSize);
  hipMalloc((void **)&coefficients_d, exponentSize);
  hipMalloc((void **)&contCounter_d, contractionsMem);
  hipMalloc((void **)&angularMoments_d, contractionsMem);
  hipMalloc((void **)&contLength_d, contractionsMem);
  hipMalloc((void **)&origin_d, originSize);
  ///////////////////////////////////////////////////////////////////////////

  ///////////////////////////////////////////////////////////////////////////
  ///CUDA copy
  hipMemcpy(primIndices_d, primIndices, totalPrim*9*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(contIndices_d, contIndices, 4*unicintegralsMem, hipMemcpyHostToDevice);
  hipMemcpy(exponents_d, exponents, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(primNormalization_d, primNormalization, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(coefficients_d, coefficients, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(contCounter_d, contCounter, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(angularMoments_d, angularMoments, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(contLength_d, contLength, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(origin_d, origin, originSize, hipMemcpyHostToDevice);
  //////////////////////////////////////////////////////////////////////////

  ////////////////////////////////////////////////////////////////////////
  ///Number of Calls to kernel

  int numberCallkernel = 0;
  
  i=0;
  int kernelIter = 0;
  int control2=0;
  while(control2<=totalPrim-1)
    {
      int control = 0;
      kernelIter = control2;
      while(control+auxNumberOfPPUC[i*3]<=totalThreads && i < unicintegrals)
	{
	  control += auxNumberOfPPUC[i*3];
          control2 += auxNumberOfPPUC[i*3];
	  i++;
	  // printf("Control: %d %d\n",i, control);
	}
      numberCallkernel++;
      integralValues = (double *)malloc(control*sizeof(double));
      hipMalloc((void **)&integralValues_d, control*sizeof(double));

      // printf("Control2: %d %d\n", numberCallkernel, control2);

           // printf("Kernel Call Number: %d\n", numberCallkernel );
      analyticInts<<<gridSize,blockSize>>>(N, primIndices_d, contIndices_d, exponents_d, primNormalization_d, coefficients_d, contCounter_d, contLength_d, origin_d, angularMoments_d, integralValues_d, control, kernelIter);

      hipMemcpy(integralValues, integralValues_d, control*sizeof(double),hipMemcpyDeviceToHost);

       for(j=kernelIter;j<control2;j++)
	{
	  integralValuesTotal[j] = integralValues[j-kernelIter];    
	  // if(numberCallkernel==3)
	     // printf("Integral post Kernel: %d, %d -> %f\n", j, j-kernelIter, integralValuesTotal[j]);
	}

      hipFree(integralValues_d);
      free(integralValues);
    }

  m=0;
  // printf("Unic Integrals Cuda:%d\n", unicintegrals);
  for(i=0; i<unicintegrals;i++)
    {
      contractedIntegrals[i] = 0.0;
      a = contIndices[i*4];
      b = contIndices[i*4+1];
      r = contIndices[i*4+2];
      s = contIndices[i*4+3];
      for(j=0; j<auxNumberOfPPUC[i*3];j++)
	{
	  contractedIntegrals[i] += contNormalization[a-1]*contNormalization[b-1]*contNormalization[r-1]*contNormalization[s-1]*integralValuesTotal[m];
	  *(contractionIntegrals+i) = contractedIntegrals[i];
	  m++;
	}
      // printf("%d %f %f %f %f\n", i, contNormalization[a],contNormalization[b],contNormalization[r],contNormalization[s]);
      printf("Contraida numero: %3d (%d,%d|%d,%d) | %15.12f \n", i,a,b,r,s,contractedIntegrals[i]);
    }

  hipFree(primIndices_d);
  hipFree(contIndices_d);
  hipFree(exponents_d);
  hipFree(primNormalization_d);
  hipFree(coefficients_d);
  hipFree(contCounter_d);
  hipFree(contLength_d);
  hipFree(origin_d);
  free(integralValuesTotal);
  free(contLength);
  free(contCounter);
  free(auxNumberOfPPUC);
  free(contIndices);
  free(exponents);
  free(primNormalization);
  free(coefficients);
  free(origin);
  free(contractedIntegrals);
  free(contNormalization);

  return;
}
